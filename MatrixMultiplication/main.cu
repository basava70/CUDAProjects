/**
 * In this project, we perform matrix multiplication
 * using CUDA. Created on August 27.
 *
 * -------------------------------------------------------*
 * Given two matrices A and B of size m *n and n * o respectively,
 * we get A * B = C which is of the size m * o.
 * The mandatory rule for the matrix multiplication is that the
 * number of columns of matrix A must match the number of rows of
 * matrix B. The size of matrix C is always number of rows of A *
 * number of columns of B.
 *
 * -------------------------------------------------------*
 * For example, consider the matrices
 *     -----------                  --------
 * A = | 1  2  3 |             B = | 7  10 |
 *     | 4  5  6 |                 | 8  11 |
 *     -----------                 | 9  12 |
 *                                 --------
 * We have C(0,0) = A[1, : ] * B[:,1]
 * C(0,0) = 1 * 7 + 2 * 8 + 3 * 9
 *
 * -------------------------------------------------------*
 * Since in C/C++, a two dimensional array or matrix is
 * represented as row major array, i.e., A = [ 1  2  3  4  5  6 ]
 * B = [ 7  10  8  11  9  12 ]. We use " j * cols + i "
 * operation to identify an element (i,j) of a matrix, i.e.,
 * A(i,j) = A[j* rows + i]; where "cols" is the number of columns of
 * the given matrix
 *
 **/

/* ---------------------------------------------------------------- */

#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

void initialize_data(int *A, int *B, int *C_cpu, int *C_gpu, const size_t row_A,
                     const size_t row_B, const size_t col_B);

void error_check(const int *C_h, const int *C_d, const size_t row_A,
                 const size_t col_B);

void print_matrix(const int *A, const size_t row_A, const size_t col_A);

void gpuMatrixMultiplication(const int *A, const int *B, int *C,
                             const size_t row_A, const size_t row_B,
                             const size_t col_B);

void gpuTiledMatrixMultiplication(const int *A, const int *B, int *C,
                                  const size_t row_A, const size_t row_B,
                                  const size_t col_B);

void cpuMatrixMultiplication(const int *A, const int *B, int *C,
                             const size_t row_A, const size_t row_B,
                             const size_t col_B);

__global__ void testKernel() {}

/* ---------------------------------------------------------------- */
int main() {

  // const size_t row_A = 1024, row_B = 2048, col_A = 2048, col_B = 728;
  // const size_t row_A = 512, row_B = 256, col_A = 256, col_B = 512;
  const size_t row_A = 1024, row_B = 2048, col_A = 2048, col_B = 1024;
  // const size_t row_A = 512, row_B = 256, col_A = 256, col_B = 512;
  // const size_t row_A = 3, row_B = 3, col_A = 3, col_B = 7;
  // const size_t row_A = 2048, row_B = 2048, col_A = 2048, col_B = 2048;

  int *A, *B, *C_cpu, *C_gpu;
  A = (int *)malloc(row_A * col_A * sizeof(int));
  B = (int *)malloc(row_B * col_B * sizeof(int));
  C_cpu = (int *)malloc(row_A * col_B * sizeof(int));
  C_gpu = (int *)malloc(row_A * col_B * sizeof(int));

  if (row_B != col_A) {
    std::cout << " row_b != col_A " << std::endl;
    exit(1);
  }

  std::cout << " ----------------------------- " << std::endl;
  std::cout << " --- Matrix Multiplication --- " << std::endl;
  std::cout << " ----------------------------- " << std::endl;
  std::cout << " A : " << row_A << " X " << col_A << std::endl;
  std::cout << " B : " << row_B << " X " << col_B << std::endl;
  std::cout << " C : " << row_A << " X " << col_B << std::endl;

  // First kernel always takes more time
  // so calling a dummy kernel before
  testKernel<<<1, 1>>>();

  initialize_data(A, B, C_cpu, C_gpu, row_A, row_B, col_B);

  // cpu matrix multiplication
  auto begin_cpu = std::chrono::high_resolution_clock::now();
  cpuMatrixMultiplication(A, B, C_cpu, row_A, row_B, col_B);
  auto end_cpu = std::chrono::high_resolution_clock::now();

  std::cout << "Time elapsed for cpu is: "
            << std::chrono::duration_cast<std::chrono::microseconds>(end_cpu -
                                                                     begin_cpu)
                       .count() /
                   1e6
            << " seconds " << std::endl;

  // Basic gpu matrix multiplication
  auto begin_gpu = std::chrono::high_resolution_clock::now();
  gpuMatrixMultiplication(A, B, C_gpu, row_A, row_B, col_B);
  auto end_gpu = std::chrono::high_resolution_clock::now();

  std::cout << "Time elapsed for basic gpu is: "
            << std::chrono::duration_cast<std::chrono::microseconds>(end_gpu -
                                                                     begin_gpu)
                       .count() /
                   1e6
            << " seconds " << std::endl;

  error_check(C_cpu, C_gpu, row_A, col_B);

  // Tiled gpu matrix multiplication
  auto begin_tiled_gpu = std::chrono::high_resolution_clock::now();
  gpuTiledMatrixMultiplication(A, B, C_gpu, row_A, row_B, col_B);
  auto end_tiled_gpu = std::chrono::high_resolution_clock::now();

  std::cout << "Time elapsed for tiled gpu is: "
            << std::chrono::duration_cast<std::chrono::microseconds>(
                   end_tiled_gpu - begin_tiled_gpu)
                       .count() /
                   1e6
            << " seconds " << std::endl;

  /* std::cout << " A =" << std::endl;
  print_matrix(A, row_A, col_A);
  std::cout << " B =" << std::endl;
  print_matrix(B, row_B, col_B);
  std::cout << " C_cpu =" << std::endl;
  print_matrix(C_cpu, row_A, col_B);
  std::cout << " C_gpu =" << std::endl;
  print_matrix(C_gpu, row_A, col_B); */

  error_check(C_cpu, C_gpu, row_A, col_B);

  std::cout << " --------------------------------- " << std::endl;
  std::cout << " ---- Error checking passed!! ---- " << std::endl;
  std::cout << " --------------------------------- " << std::endl;

  // free the variables
  free(A);
  free(B);
  free(C_gpu);
  free(C_cpu);

  return 0;
}
