/**
 * In this project, we perform matrix multiplication
 * using CUDA. Created on August 27.
 *
 * -------------------------------------------------------*
 * Given two matrices A and B of size m *n and n * o respectively,
 * we get A * B = C which is of the size m * o.
 * The mandatory rule for the matrix multiplication is that the
 * number of columns of matrix A must match the number of rows of
 * matrix B. The size of matrix C is always number of rows of A *
 * number of columns of B.
 *
 * -------------------------------------------------------*
 * For example, consider the matrices
 *     -----------                  --------
 * A = | 1  2  3 |             B = | 7  10 |
 *     | 4  5  6 |                 | 8  11 |
 *     -----------                 | 9  12 |
 *                                 --------
 * We have C(0,0) = A[1, : ] * B[:,1]
 * C(0,0) = 1 * 7 + 2 * 8 + 3 * 9
 *
 * -------------------------------------------------------*
 * Since in C/C++, a two dimensional array or matrix is
 * represented as row major array, i.e., A = [ 1  2  3  4  5  6 ]
 * B = [ 7  10  8  11  9  12 ]. We use " j * cols + i "
 * operation to identify an element (i,j) of a matrix, i.e.,
 * A(i,j) = A[j* rows + i]; where "cols" is the number of columns of
 * the given matrix
 *
 **/


#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdlib>
#include <iostream>

void initialize_data(int *A, int *B, int *C_cpu, int *C_gpu, const size_t row_A,
                     const size_t row_B, const size_t col_B);

void error_check(const int *C_h, const int *C_d, const size_t row_A,
                 const size_t col_B);

void print_matrix(const int *A, const size_t row_A, const size_t col_A);

void gpuMatrixMultiplication(const int *A, const int *B, int *C,
                             const size_t row_A, const size_t row_B,
                             const size_t col_B);

void cpuMatrixMultiplication(const int *A, const int *B, int *C,
                             const size_t row_A, const size_t row_B,
                             const size_t col_B);

int main() {

  const size_t row_A = 1024, row_B = 2048, col_A = 2048, col_B = 1024;

  int *A, *B, *C_cpu, *C_gpu;
  A = (int *)malloc(row_A * col_A * sizeof(int));
  B = (int *)malloc(row_B * col_B * sizeof(int));
  C_cpu = (int *)malloc(row_A * col_B * sizeof(int));
  C_gpu = (int *)malloc(row_A * col_B * sizeof(int));

  if (row_B != col_A) {
    std::cout << " row_b != col_A " << std::endl;
    exit(1);
  }

  std::cout << " ----------------------------- " << std::endl;
  std::cout << " --- Matrix Multiplication --- " << std::endl;
  std::cout << " ----------------------------- " << std::endl;
  std::cout << " row_A : " << row_A << " col_A : " << col_A << std::endl;
  std::cout << " row_B : " << row_B << " col_B : " << col_B << std::endl;
  std::cout << " row_C : " << row_A << " col_C : " << col_B << std::endl;

  initialize_data(A, B, C_cpu, C_gpu, row_A, row_B, col_B);

  auto begin_gpu = std::chrono::high_resolution_clock::now();
  gpuMatrixMultiplication(A, B, C_gpu, row_A, row_B, col_B);
  auto end_gpu = std::chrono::high_resolution_clock::now();

  std::cout << "Time elapsed for gpu is: "
            << std::chrono::duration_cast<std::chrono::microseconds>(end_gpu -
                                                                     begin_gpu)
                       .count() /
                   1e6
            << " seconds " << std::endl;

  auto begin_cpu = std::chrono::high_resolution_clock::now();
  cpuMatrixMultiplication(A, B, C_cpu, row_A, row_B, col_B);
  auto end_cpu = std::chrono::high_resolution_clock::now();

  std::cout << "Time elapsed for cpu is: "
            << std::chrono::duration_cast<std::chrono::microseconds>(end_cpu -
                                                                     begin_cpu)
                       .count() /
                   1e6
            << " seconds " << std::endl;

  /* std::cout << " A =" << std::endl;
  print_matrix(A, row_A, col_A);
  std::cout << " B =" << std::endl;
  print_matrix(B, row_B, col_B);
  std::cout << " C_cpu =" << std::endl;
  print_matrix(C_cpu, row_A, col_B);
  std::cout << " C_gpu =" << std::endl;
  print_matrix(C_gpu, row_A, col_B); */

  error_check(C_cpu, C_gpu, row_A, col_B);

  std::cout << " ------------------------ " << std::endl;
  std::cout << " ------- Success -------- " << std::endl;
  std::cout << " ------------------------ " << std::endl;

  // free the variables
  free(A);
  free(B);
  free(C_gpu);
  free(C_cpu);

  return 0;
}
